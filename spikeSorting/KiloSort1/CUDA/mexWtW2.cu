#include "hip/hip_runtime.h"
/*
 * Example of how to use the mxGPUArray API in a MEX file.  This example shows
 * how to write a MEX function that takes a gpuArray input and returns a
 * gpuArray output, e.g. B=mexFunction(A).
 *
 * Copyright 2012 The MathWorks, Inc.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <cstdlib>
#include <algorithm>
#include <iostream>
using namespace std;

const int Nthreads = 1024,   nblock = 32;
//////////////////////////////////////////////////////////////////////////////////////////

__global__ void	crossFilter(const double *Params, const float *W1, const float *W2,
        const float *UtU, float *WtW){    
  __shared__ float shW1[nblock*81], shW2[nblock*81]; 

  float x;
  int nt0, tidx, tidy , bidx, bidy, i, NT, Nfilt, t;

  tidx 		= threadIdx.x;
  tidy 		= threadIdx.y;
  bidx 		= blockIdx.x;
  bidy 		= blockIdx.y;
  
  Nfilt = (int) Params[1];
  nt0       = (int) Params[9];
  
  while(tidx<nt0){
    shW1[tidx + tidy * nt0] = W1[tidx + (tidy+bidx*nblock) * nt0];
    shW2[tidx + tidy * nt0] = W2[tidx + (tidy+bidy*nblock) * nt0];
    tidx+= nblock;
  }
  tidx 		= threadIdx.x;
  __syncthreads();
	 	 
  for(i=0;i<2*nt0-1;i++){
      x = 0.0f;
      if(i<nt0)
          for(t=0;t<i+1;t++)
              x += shW1[t + nt0 * tidx] * shW2[t + (nt0-i-1) + nt0 * tidy];
      else
          for(t=i-nt0+1;t<nt0;t++)
              x += shW1[t + nt0 * tidx] * shW2[t + (nt0-i-1) + nt0 * tidy];
      WtW[tidx+bidx*nblock + (tidy + bidy*nblock)*Nfilt +  i*Nfilt*Nfilt] =
              x * UtU[tidx+bidx*nblock + (tidy + bidy*nblock)*Nfilt];
  }
}


//////////////////////////////////////////////////////////////////////////////////////////

/*
 * Host code
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
    /* Declare input variables*/
  double *Params, *d_Params;
  int nt0, Nfilt, NT;

  /* Initialize the MathWorks GPU API. */
  mxInitGPU();

  /* read Params and copy to GPU */
  Params  	= (double*) mxGetData(prhs[0]);
  NT		= (int) Params[0];
  Nfilt		= (int) Params[1];
  nt0       = (int) Params[9];
  
  hipMalloc(&d_Params,      sizeof(double)*mxGetNumberOfElements(prhs[0]));
  hipMemcpy(d_Params,Params,sizeof(double)*mxGetNumberOfElements(prhs[0]),hipMemcpyHostToDevice);

  /* collect input GPU variables*/
  mxGPUArray const  *W1, *W2,   *UtU;
  const float     *d_W1,*d_W2, *d_UtU;
  
  W1             = mxGPUCreateFromMxArray(prhs[1]);
  d_W1        	= (float const *)(mxGPUGetDataReadOnly(W1));
  W2             = mxGPUCreateFromMxArray(prhs[2]);
  d_W2        	= (float const *)(mxGPUGetDataReadOnly(W2));
  UtU       	= mxGPUCreateFromMxArray(prhs[3]);
  d_UtU     	= (float const *)(mxGPUGetDataReadOnly(UtU));


  mxGPUArray *WtW;
  float  *d_WtW;
  const mwSize dimsu[] 	= {Nfilt, Nfilt, 2*nt0-1}; 
  WtW 		= mxGPUCreateGPUArray(3, dimsu, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);  
  d_WtW 		= (float *)(mxGPUGetData(WtW));

  dim3 grid(Nfilt/nblock, Nfilt/nblock);
  dim3 block(nblock, nblock);
  crossFilter<<<grid, block>>>(d_Params, d_W1, d_W2, d_UtU, d_WtW); 

  plhs[0] 	= mxGPUCreateMxArrayOnGPU(WtW);

  hipFree(d_Params);
  mxGPUDestroyGPUArray(WtW);
  mxGPUDestroyGPUArray(W1);
  mxGPUDestroyGPUArray(W2);
  mxGPUDestroyGPUArray(UtU);
  
}
